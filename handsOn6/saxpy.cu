#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void saxpy(int n, float *x, float *y){
int i = threadIdx.x;
if(i < n)
y[i] = x[i] + y[i];
}

void printVector(float *vector, int n)
{

 for (int i=0; i < n ; ++i)
  printf("%1.0f\t", vector[i]);

  printf("\n\n");
}

void generateVector(float *vector, int n)
{
 for (int i=0; i < n ; ++i)
  vector[i] = i + 1;
}

int main(int argc, char *argv[])
{
  int n = atoi(argv[1]);   
  float *x,*y;

  x = (float*) malloc(sizeof(float) * n);
  y = (float*) malloc(sizeof(float) * n);
  
  generateVector(x, n);
  printVector(x, n);

  generateVector(y, n);
  printVector(y, n);
  
  float *xd, *yd;
  
  hipMalloc( (void**)&xd, sizeof(float) * n );
  hipMalloc( (void**)&yd, sizeof(float) * n );
  
  hipMemcpy(xd, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(yd, y, sizeof(float) * n, hipMemcpyHostToDevice);
  
  int NUMBER_OF_BLOCKS = 1;
  int NUMBER_OF_THREADS_PER_BLOCK = n;
  saxpy<<< NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK >>>(n, xd, yd);
  
  hipDeviceSynchronize();
  
  hipMemcpy(y, yd, sizeof(float) * (n), hipMemcpyDeviceToHost);
  printVector(y, n);
  
  hipFree(xd);
  hipFree(yd);
  
  free(x);
  free(y);
 
  return 0;

}
